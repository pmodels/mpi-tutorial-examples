#include "hip/hip_runtime.h"
/* -*- Mode: C; c-basic-offset:4 ; indent-tabs-mode:nil ; -*- */
/*
 * See COPYRIGHT in top-level directory.
 */

/*
 * 2D stencil code using a nonblocking send/receive with manual packing/unpacking.
 *
 * 2D regular grid is divided into px * py blocks of grid points (px * py = # of processes.)
 * In every iteration, each process calls nonblocking operations to exchange a halo with
 * neighbors. Grid points in a halo are packed and unpacked before and after communications.
 */

#include "stencil_par.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

/* Comment out if MPI library does not support GPUDirect */
#define HAVE_GPUDIRECT

/* CUDA allows a maximum of 1024 threads per block */
#define CUDA_BLKSIZE_X 32
#define CUDA_BLKSIZE_Y 32

#define MIN(a, b) ((a < b) ? a : b)

/* row-major order */
#define ind(i,j) ((j)*(bx+2)+(i))

void setup(int rank, int proc, int argc, char **argv, int *n_ptr, int *energy_ptr, int *niters_ptr,
           int *px_ptr, int *py_ptr, int *final_flag);

void init_sources(int bx, int by, int offx, int offy, int n, const int nsources, int sources[][2],
                  int *locnsources_ptr, int locsources[][2]);

void alloc_bufs(int bx, int by, double **aold_ptr, double **anew_ptr, double **sbufnorth_ptr,
                double **sbufsouth_ptr, double **sbufeast_ptr, double **sbufwest_ptr,
                double **rbufnorth_ptr, double **rbufsouth_ptr, double **rbufeast_ptr,
                double **rbufwest_ptr);

void alloc_dev_bufs(int bx, int by, double **aold_ptr, double **anew_ptr, double **sbufnorth_ptr,
                    double **sbufsouth_ptr, double **sbufeast_ptr, double **sbufwest_ptr,
                    double **rbufnorth_ptr, double **rbufsouth_ptr, double **rbufeast_ptr,
                    double **rbufwest_ptr);

void free_bufs(double *aold, double *anew, double *sbufnorth, double *sbufsouth, double *sbufeast,
               double *sbufwest, double *rbufnorth, double *rbufsouth, double *rbufeast,
               double *rbufwest);

void free_dev_bufs(double *aold, double *anew, double *sbufnorth, double *sbufsouth,
                   double *sbufeast, double *sbufwest, double *rbufnorth, double *rbufsouth,
                   double *rbufeast, double *rbufwest);

__global__ void update_sources(int bx, int by, double heat, int nsources, int *locsources,
                               double *aold);

__global__ void pack_data(int bx, int by, double *aold, double *sbufnorth, double *sbufsouth,
                          double *sbfueast, double *sbufwest);

__global__ void unpack_data(int bx, int by, double *aold, double *rbufnorth, double *rbufsouth,
                            double *rbufeast, double *rbufwest);

__global__ void update_grid(int bx, int by, double *aold, double *anew, double *heat_ptr);

int main(int argc, char **argv)
{
    int rank, size;
    int n, energy, niters, px, py;

    int rx, ry;
    int north, south, west, east;
    int bx, by, offx, offy;

    /* three heat sources */
    int sources[NSOURCES][2];
    int locnsources;            /* number of sources in my area */
    int locsources[NSOURCES][2];        /* sources local to my rank */
    int *locsources_d;

    double t1, t2;

    int iter, i;

#ifndef HAVE_GPUDIRECT
    /* host buffers */
    double *sbufnorth, *sbufsouth, *sbufeast, *sbufwest;
    double *rbufnorth, *rbufsouth, *rbufeast, *rbufwest;
    double *aold, *anew;
#endif

    /* device buffers */
    double *sbufnorth_d, *sbufsouth_d, *sbufeast_d, *sbufwest_d;
    double *rbufnorth_d, *rbufsouth_d, *rbufeast_d, *rbufwest_d;
    double *aold_d, *anew_d, *tmp;

    double heat = 0.0, rheat;
    double *heat_d;

    int final_flag;

    /* assign device to process */
    int local_rank, dev_id, dev_count;
    local_rank = atoi(getenv("PMI_RANK"));
    hipGetDeviceCount(&dev_count);
    dev_id = local_rank % dev_count;
    hipSetDevice(dev_id);

    /* initialize MPI envrionment */
    MPI_Init(&argc, &argv);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &size);

    /* argument checking and setting */
    setup(rank, size, argc, argv, &n, &energy, &niters, &px, &py, &final_flag);

    if (final_flag == 1) {
        MPI_Finalize();
        exit(0);
    }

    /* determine my coordinates (x,y) -- rank=x*a+y in the 2d processor array */
    rx = rank % px;
    ry = rank / px;

    /* determine my four neighbors */
    north = (ry - 1) * px + rx;
    if (ry - 1 < 0)
        north = MPI_PROC_NULL;
    south = (ry + 1) * px + rx;
    if (ry + 1 >= py)
        south = MPI_PROC_NULL;
    west = ry * px + rx - 1;
    if (rx - 1 < 0)
        west = MPI_PROC_NULL;
    east = ry * px + rx + 1;
    if (rx + 1 >= px)
        east = MPI_PROC_NULL;

    /* decompose the domain */
    bx = n / px;        /* block size in x */
    by = n / py;        /* block size in y */
    offx = rx * bx;     /* offset in x */
    offy = ry * by;     /* offset in y */

    /* printf("%i (%i,%i) - w: %i, e: %i, n: %i, s: %i\n", rank, ry,rx,west,east,north,south); */

    /* initialize three heat sources */
    init_sources(bx, by, offx, offy, n, NSOURCES, sources, &locnsources, locsources);

    /* create an asynchronous cuda stream */
    hipStream_t s;
    hipStreamCreateWithFlags(&s, hipStreamNonBlocking);

    /* allocate space for params in device memory */
    hipMalloc(&locsources_d, sizeof(int) * nsources * 2);

    /* move parameters to device memory */
    hipMemcpy(locsources_d, locsources, sizeof(int) * nsources * 2, hipMemcpyHostToDevice);

#ifndef HAVE_GPUDIRECT
    /* allocate working arrays & communication buffers */
    alloc_bufs(bx, by, &aold, &anew,
               &sbufnorth, &sbufsouth, &sbufeast, &sbufwest,
               &rbufnorth, &rbufsouth, &rbufeast, &rbufwest);
#endif

    /* allocate working arrays & communication buffers for device */
    alloc_dev_bufs(bx, by, &aold_d, &anew_d,
                   &sbufnorth_d, &sbufsouth_d, &sbufeast_d, &sbufwest_d,
                   &rbufnorth_d, &rbufsouth_d, &rbufeast_d, &rbufwest_d);

    /* cuda kernels execution configuration parameters */
    int cuGrdSzX, cuGrdSzY, cuBlkSzX, cuBlkSzY;
    int cuGrdSzPackUnpack, cuBlkSzPackUnpack;

    /* workout number of threads in block for each dimension */
    cuBlkSzX = (bx > CUDA_BLKSIZE_X) ? CUDA_BLKSIZE_X : bx;
    cuBlkSzY = (by > CUDA_BLKSIZE_Y) ? CUDA_BLKSIZE_Y : by;
    dim3 cuBlkSzUpdate(cuBlkSzX, cuBlkSzY);

    /* workout the size of shared memory for heat reduction */
    size_t shmSize = (cuBlkSzX * cuBlkSzY * sizeof(double));

    /* workout number of blocks in grid for each dimension */
    cuGrdSzX = (bx + (cuBlkSzX - 1)) / cuBlkSzX;
    cuGrdSzY = (by + (cuBlkSzY - 1)) / cuBlkSzY;
    dim3 cuGrdSzUpdate(cuGrdSzX, cuGrdSzY);

    /* workout number of threads in block for each pack/unpack */
    int bmax = (bx > by) ? bx : by;
    cuBlkSzPackUnpack =
        (bmax > (CUDA_BLKSIZE_X * CUDA_BLKSIZE_Y)) ? (CUDA_BLKSIZE_X * CUDA_BLKSIZE_Y) : bmax;

    /* workout number of blocks in grid for each pack/unpack */
    cuGrdSzPackUnpack = (bmax + (cuBlkSzPackUnpack - 1)) / cuBlkSzPackUnpack;

    /* allocate heat vector for reduction: threads inside the same block update
     * heat values in shared memory and reflect those updates in global memory,
     * eventually.
     *
     * Rationale: heat values cannot be reduced directly by the same kernel as
     * thread synchronization across thread blocks is not easy task. For this
     * reason we use a temp array to store partial heat updates and run the
     * reduction in the host at the end of the program instead. */
    hipMalloc(&heat_d, sizeof(double) * cuGrdSzX * cuGrdSzY);
    hipMemset(heat_d, 0, sizeof(double) * cuGrdSzX * cuGrdSzY);

    t1 = MPI_Wtime();   /* take time */

    for (iter = 0; iter < niters; ++iter) {

        /* refresh heat sources */
        update_sources <<< cuGrdSzUpdate, cuBlkSzUpdate, 0, s >>> (bx, by, energy, nsources,
                                                                   locsources_d, aold_d);

        /* pack data in device */
        pack_data <<< cuGrdSzPackUnpack, cuBlkSzPackUnpack, 0, s >>> (bx, by, aold_d,
                                                                      sbufnorth_d, sbufsouth_d,
                                                                      sbufeast_d, sbufwest_d);

        /* wait for stream operations to complete */
        hipStreamSynchronize(s);

        MPI_Request reqs[8];

#ifdef HAVE_GPUDIRECT
        /* exchange data with neighbors */
        MPI_Isend(sbufnorth_d, bx, MPI_DOUBLE, north, 9, MPI_COMM_WORLD, &reqs[0]);
        MPI_Isend(sbufsouth_d, bx, MPI_DOUBLE, south, 9, MPI_COMM_WORLD, &reqs[1]);
        MPI_Isend(sbufeast_d, by, MPI_DOUBLE, east, 9, MPI_COMM_WORLD, &reqs[2]);
        MPI_Isend(sbufwest_d, by, MPI_DOUBLE, west, 9, MPI_COMM_WORLD, &reqs[3]);

        MPI_Irecv(rbufnorth_d, bx, MPI_DOUBLE, north, 9, MPI_COMM_WORLD, &reqs[4]);
        MPI_Irecv(rbufsouth_d, bx, MPI_DOUBLE, south, 9, MPI_COMM_WORLD, &reqs[5]);
        MPI_Irecv(rbufeast_d, by, MPI_DOUBLE, east, 9, MPI_COMM_WORLD, &reqs[6]);
        MPI_Irecv(rbufwest_d, by, MPI_DOUBLE, west, 9, MPI_COMM_WORLD, &reqs[7]);

        MPI_Waitall(8, reqs, MPI_STATUSES_IGNORE);
#else
        /* move data to host */
        hipMemcpy(sbufnorth, sbufnorth_d, sizeof(double) * bx, hipMemcpyDeviceToHost);
        hipMemcpy(sbufsouth, sbufsouth_d, sizeof(double) * bx, hipMemcpyDeviceToHost);
        hipMemcpy(sbufeast, sbufeast_d, sizeof(double) * bx, hipMemcpyDeviceToHost);
        hipMemcpy(sbufwest, sbufwest_d, sizeof(double) * bx, hipMemcpyDeviceToHost);

        /* exchange data with neighbors */
        MPI_Isend(sbufnorth, bx, MPI_DOUBLE, north, 9, MPI_COMM_WORLD, &reqs[0]);
        MPI_Isend(sbufsouth, bx, MPI_DOUBLE, south, 9, MPI_COMM_WORLD, &reqs[1]);
        MPI_Isend(sbufeast, by, MPI_DOUBLE, east, 9, MPI_COMM_WORLD, &reqs[2]);
        MPI_Isend(sbufwest, by, MPI_DOUBLE, west, 9, MPI_COMM_WORLD, &reqs[3]);

        MPI_Irecv(rbufnorth, bx, MPI_DOUBLE, north, 9, MPI_COMM_WORLD, &reqs[4]);
        MPI_Irecv(rbufsouth, bx, MPI_DOUBLE, south, 9, MPI_COMM_WORLD, &reqs[5]);
        MPI_Irecv(rbufeast, by, MPI_DOUBLE, east, 9, MPI_COMM_WORLD, &reqs[6]);
        MPI_Irecv(rbufwest, by, MPI_DOUBLE, west, 9, MPI_COMM_WORLD, &reqs[7]);

        MPI_Waitall(8, reqs, MPI_STATUSES_IGNORE);

        /* move received data to device */
        hipMemcpy(rbufnorth_d, rbufnorth, sizeof(double) * bx, hipMemcpyHostToDevice);
        hipMemcpy(rbufsouth_d, rbufsouth, sizeof(double) * bx, hipMemcpyHostToDevice);
        hipMemcpy(rbufeast_d, rbufeast, sizeof(double) * bx, hipMemcpyHostToDevice);
        hipMemcpy(rbufwest_d, rbufwest, sizeof(double) * bx, hipMemcpyHostToDevice);
#endif

        /* unpack data in device */
        unpack_data <<< cuGrdSzPackUnpack, cuBlkSzPackUnpack, 0, s >>> (bx, by,
                                                                        aold_d, rbufnorth_d,
                                                                        rbufsouth_d, rbufeast_d,
                                                                        rbufwest_d);

        /* update grid points */
        update_grid <<< cuGrdSzUpdate, cuBlkSzUpdate, shmSize, s >>> (bx, by, aold_d, anew_d,
                                                                      heat_d);

        /* swap working arrays */
        tmp = anew_d;
        anew_d = aold_d;
        aold_d = tmp;
    }

    /* wait for kernel updates to grid */
    hipStreamSynchronize(s);

    t2 = MPI_Wtime();   /* take time */

    /* reduce partial heat updates in host */
    double *heat_h = (double *) malloc(sizeof(double) * cuGrdSzX * cuGrdSzY);
    hipMemcpy(heat_h, heat_d, sizeof(double) * cuGrdSzX * cuGrdSzY, hipMemcpyDeviceToHost);
    for (i = 0; i < cuGrdSzX * cuGrdSzY; i++)
        heat += heat_h[i];

    /* get final heat in the system */
    MPI_Allreduce(&heat, &rheat, 1, MPI_DOUBLE, MPI_SUM, MPI_COMM_WORLD);
    if (rank == 0)
        printf("[%i] last heat: %f time: %f\n", rank, rheat, t2 - t1);

#ifndef HAVE_GPUDIRECT
    /* free working arrays and communication buffers */
    free_bufs(aold, anew, sbufnorth, sbufsouth, sbufeast, sbufwest,
              rbufnorth, rbufsouth, rbufeast, rbufwest);
#endif

    /* free working device arrays and communication buffers */
    free_dev_bufs(aold_d, anew_d, sbufnorth_d, sbufsouth_d, sbufeast_d, sbufwest_d,
                  rbufnorth_d, rbufsouth_d, rbufeast_d, rbufwest_d);

    /* free parameters in device memory */
    hipFree(locsources_d);
    hipFree(heat_d);
    free(heat_h);

    hipStreamDestroy(s);

    MPI_Finalize();
    return 0;
}

void setup(int rank, int proc, int argc, char **argv,
           int *n_ptr, int *energy_ptr, int *niters_ptr, int *px_ptr, int *py_ptr, int *final_flag)
{
    int n, energy, niters, px, py;

    (*final_flag) = 0;

    if (argc < 6) {
        if (rank == 0)
            printf("usage: stencil_mpi <n> <energy> <niters> <px> <py>\n");
        (*final_flag) = 1;
        return;
    }

    n = atoi(argv[1]);  /* nxn grid */
    energy = atoi(argv[2]);     /* energy to be injected per iteration */
    niters = atoi(argv[3]);     /* number of iterations */
    px = atoi(argv[4]); /* 1st dim processes */
    py = atoi(argv[5]); /* 2nd dim processes */

    if (px * py != proc) {
        fprintf(stderr, "px * py must equal to the number of processes.\n");
        MPI_Abort(MPI_COMM_WORLD, 1);   /* abort if px or py are wrong */
    }
    if (n % px != 0) {
        fprintf(stderr, "grid size n must be divisible by px.\n");
        MPI_Abort(MPI_COMM_WORLD, 2);   /* abort px needs to divide n */
    }
    if (n % py != 0) {
        fprintf(stderr, "grid size n must be divisible by py.\n");
        MPI_Abort(MPI_COMM_WORLD, 3);   /* abort py needs to divide n */
    }

    (*n_ptr) = n;
    (*energy_ptr) = energy;
    (*niters_ptr) = niters;
    (*px_ptr) = px;
    (*py_ptr) = py;
}

void init_sources(int bx, int by, int offx, int offy, int n,
                  const int nsources, int sources[][2], int *locnsources_ptr, int locsources[][2])
{
    int i, locnsources = 0;

    sources[0][0] = n / 2;
    sources[0][1] = n / 2;
    sources[1][0] = n / 3;
    sources[1][1] = n / 3;
    sources[2][0] = n * 4 / 5;
    sources[2][1] = n * 8 / 9;

    for (i = 0; i < nsources; ++i) {    /* determine which sources are in my patch */
        int locx = sources[i][0] - offx;
        int locy = sources[i][1] - offy;
        if (locx >= 0 && locx < bx && locy >= 0 && locy < by) {
            locsources[locnsources][0] = locx + 1;      /* offset by halo zone */
            locsources[locnsources][1] = locy + 1;      /* offset by halo zone */
            locnsources++;
        }
    }

    (*locnsources_ptr) = locnsources;
}

void alloc_bufs(int bx, int by, double **aold_ptr, double **anew_ptr,
                double **sbufnorth_ptr, double **sbufsouth_ptr,
                double **sbufeast_ptr, double **sbufwest_ptr,
                double **rbufnorth_ptr, double **rbufsouth_ptr,
                double **rbufeast_ptr, double **rbufwest_ptr)
{
    double *aold, *anew;
    double *sbufnorth, *sbufsouth, *sbufeast, *sbufwest;
    double *rbufnorth, *rbufsouth, *rbufeast, *rbufwest;

    /* allocate two working arrays */
    anew = (double *) malloc((bx + 2) * (by + 2) * sizeof(double));     /* 1-wide halo zones! */
    aold = (double *) malloc((bx + 2) * (by + 2) * sizeof(double));     /* 1-wide halo zones! */

    memset(aold, 0, (bx + 2) * (by + 2) * sizeof(double));
    memset(anew, 0, (bx + 2) * (by + 2) * sizeof(double));

    /* allocate communication buffers */
    sbufnorth = (double *) malloc(bx * sizeof(double)); /* send buffers */
    sbufsouth = (double *) malloc(bx * sizeof(double));
    sbufeast = (double *) malloc(by * sizeof(double));
    sbufwest = (double *) malloc(by * sizeof(double));
    rbufnorth = (double *) malloc(bx * sizeof(double)); /* receive buffers */
    rbufsouth = (double *) malloc(bx * sizeof(double));
    rbufeast = (double *) malloc(by * sizeof(double));
    rbufwest = (double *) malloc(by * sizeof(double));

    memset(sbufnorth, 0, bx * sizeof(double));
    memset(sbufsouth, 0, bx * sizeof(double));
    memset(sbufeast, 0, by * sizeof(double));
    memset(sbufwest, 0, by * sizeof(double));
    memset(rbufnorth, 0, bx * sizeof(double));
    memset(rbufsouth, 0, bx * sizeof(double));
    memset(rbufeast, 0, by * sizeof(double));
    memset(rbufwest, 0, by * sizeof(double));

    (*aold_ptr) = aold;
    (*anew_ptr) = anew;
    (*sbufnorth_ptr) = sbufnorth;
    (*sbufsouth_ptr) = sbufsouth;
    (*sbufeast_ptr) = sbufeast;
    (*sbufwest_ptr) = sbufwest;
    (*rbufnorth_ptr) = rbufnorth;
    (*rbufsouth_ptr) = rbufsouth;
    (*rbufeast_ptr) = rbufeast;
    (*rbufwest_ptr) = rbufwest;
}

void alloc_dev_bufs(int bx, int by, double **aold_ptr, double **anew_ptr,
                    double **sbufnorth_ptr, double **sbufsouth_ptr,
                    double **sbufeast_ptr, double **sbufwest_ptr,
                    double **rbufnorth_ptr, double **rbufsouth_ptr,
                    double **rbufeast_ptr, double **rbufwest_ptr)
{
    double *aold, *anew;
    double *sbufnorth, *sbufsouth, *sbufeast, *sbufwest;
    double *rbufnorth, *rbufsouth, *rbufeast, *rbufwest;

    /* allocate two working arrays */
    hipMalloc(&anew, ((bx + 2) * (by + 2)) * sizeof(double));  /* 1-wide halo zones! */
    hipMalloc(&aold, ((bx + 2) * (by + 2)) * sizeof(double));  /* 1-wide halo zones! */

    hipMemset(anew, 0, ((bx + 2) * (by + 2)) * sizeof(double));
    hipMemset(aold, 0, ((bx + 2) * (by + 2)) * sizeof(double));

    /* allocate communication buffers */
    hipMalloc(&sbufnorth, bx * sizeof(double));
    hipMalloc(&sbufsouth, bx * sizeof(double));
    hipMalloc(&sbufeast, by * sizeof(double));
    hipMalloc(&sbufwest, by * sizeof(double));
    hipMalloc(&rbufnorth, bx * sizeof(double));
    hipMalloc(&rbufsouth, bx * sizeof(double));
    hipMalloc(&rbufeast, by * sizeof(double));
    hipMalloc(&rbufwest, by * sizeof(double));

    hipMemset(sbufnorth, 0, bx * sizeof(double));
    hipMemset(sbufsouth, 0, bx * sizeof(double));
    hipMemset(sbufeast, 0, by * sizeof(double));
    hipMemset(sbufwest, 0, by * sizeof(double));
    hipMemset(rbufnorth, 0, bx * sizeof(double));
    hipMemset(rbufsouth, 0, bx * sizeof(double));
    hipMemset(rbufeast, 0, by * sizeof(double));
    hipMemset(rbufwest, 0, by * sizeof(double));

    (*aold_ptr) = aold;
    (*anew_ptr) = anew;
    (*sbufnorth_ptr) = sbufnorth;
    (*sbufsouth_ptr) = sbufsouth;
    (*sbufeast_ptr) = sbufeast;
    (*sbufwest_ptr) = sbufwest;
    (*rbufnorth_ptr) = rbufnorth;
    (*rbufsouth_ptr) = rbufsouth;
    (*rbufeast_ptr) = rbufeast;
    (*rbufwest_ptr) = rbufwest;
}

void free_bufs(double *aold, double *anew,
               double *sbufnorth, double *sbufsouth,
               double *sbufeast, double *sbufwest,
               double *rbufnorth, double *rbufsouth, double *rbufeast, double *rbufwest)
{
    free(aold);
    free(anew);
    free(sbufnorth);
    free(sbufsouth);
    free(sbufeast);
    free(sbufwest);
    free(rbufnorth);
    free(rbufsouth);
    free(rbufeast);
    free(rbufwest);
}

void free_dev_bufs(double *aold, double *anew,
                   double *sbufnorth, double *sbufsouth,
                   double *sbufeast, double *sbufwest,
                   double *rbufnorth, double *rbufsouth, double *rbufeast, double *rbufwest)
{
    hipFree(aold);
    hipFree(anew);
    hipFree(sbufnorth);
    hipFree(sbufsouth);
    hipFree(sbufeast);
    hipFree(sbufwest);
    hipFree(rbufnorth);
    hipFree(rbufsouth);
    hipFree(rbufeast);
    hipFree(rbufwest);
}

__global__ void update_sources(int bx, int by, double heat, int nsources, int *locsources,
                               double *aold)
{
    int (*locsources_)[2] = (int (*)[2]) locsources;
    int i = 1 + (blockIdx.x * blockDim.x) + threadIdx.x;
    int j = 1 + (blockIdx.y * blockDim.y) + threadIdx.y;

    /* A kernel for updating a few doubles in the grid is overkill
     * and ideally should be done during update of the grid. However,
     * for sake of demonstration this should be fine. */
    if (i < (bx + 1) && j < (by + 1)) {
        for (int k = 0; k < nsources; k++) {
            if (i == locsources_[k][0] && j == locsources_[k][1])
                aold[ind(i, j)] += heat;
        }
    }
}

__global__ void pack_data(int bx, int by, double *aold,
                          double *sbufnorth, double *sbufsouth, double *sbufeast, double *sbufwest)
{
    /* Get thread idx in global domain */
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;

    /* First do the north buf packing */
    if (i < bx)
        sbufnorth[i] = aold[ind(i + 1, 1)];

    /* Next do the south buf packing */
    if (i < bx)
        sbufsouth[i] = aold[ind(i + 1, by)];

    /* Then do the east buf packing */
    if (i < by)
        sbufeast[i] = aold[ind(bx, i + 1)];

    /* Finally do the west buf packing */
    if (i < by)
        sbufwest[i] = aold[ind(1, i + 1)];
}

__global__ void unpack_data(int bx, int by, double *aold,
                            double *rbufnorth, double *rbufsouth, double *rbufeast,
                            double *rbufwest)
{
    /* Get thread idx in global domain */
    int i = (blockIdx.x * blockDim.x) + threadIdx.x;

    /* First do the north buf unpacking */
    if (i < bx)
        aold[ind(i + 1, 0)] = rbufnorth[i];

    /* Next do the south buf unpacking */
    if (i < bx)
        aold[ind(i + 1, by + 1)] = rbufsouth[i];

    /* Then do the east buf unpacking */
    if (i < by)
        aold[ind(bx + 1, i + 1)] = rbufeast[i];

    /* Finally do the west buf unpacking */
    if (i < by)
        aold[ind(0, i + 1)] = rbufwest[i];
}

__global__ void update_grid(int bx, int by, double *aold, double *anew, double *heat)
{
    /* Shared memory size specified at kernel launch */
    extern __shared__ double heat_[];
    int blkDimX, blkDimY;

    /* Calculate index in matrix for thread in threadblock */
    int i = 1 + (blockDim.x * blockIdx.x) + threadIdx.x;
    int j = 1 + (blockDim.y * blockIdx.y) + threadIdx.y;
    int ii = threadIdx.x;
    int jj = threadIdx.y;
    blkDimX = MIN(blockDim.x, bx - (blockDim.x * blockIdx.x));

    /* Update heat value at thread location */
    if (i < (bx + 1) && j < (by + 1)) {
        anew[ind(i, j)] =
            aold[ind(i, j)] / 2.0 + (aold[ind(i - 1, j)] +
                                     aold[ind(i + 1, j)] +
                                     aold[ind(i, j - 1)] + aold[ind(i, j + 1)]) / 4.0 / 2.0;

        /* store partial heat values in shared memory */
        heat_[(jj * blkDimX) + ii] = anew[ind(i, j)];
    }

    /* Wait for all threads in the block to complete */
    __syncthreads();

    /* First thread in threadblock reduces heat values into global memory:
     * could be done more efficiently with a hierarchical reduction; Again,
     * here we don't care about performance. */
    if (ii == 0 && jj == 0) {
        /* account for cases in which the thread block is not multiple of
         * bx and/or by */
        blkDimX = MIN(blockDim.x, bx - (blockDim.x * blockIdx.x));
        blkDimY = MIN(blockDim.y, by - (blockDim.y * blockIdx.y));
        double reduce = 0.0;
        for (j = 0; j < blkDimY; j++)
            for (i = 0; i < blkDimX; i++)
                reduce += heat_[(j * blkDimX) + i];
        heat[(blockIdx.y * gridDim.x) + blockIdx.x] = reduce;
    }
}
